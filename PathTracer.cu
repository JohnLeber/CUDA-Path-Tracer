#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <Windows.h>//needed to save output as a bitmap file.
#include <stdio.h>
#include "PathTracer.h"
#include "PTProgress.h"
//#include <thrust/random.h>
#include <hiprand/hiprand_kernel.h>
//#include <thrust/random.h>
// 
//--------------------------------------------------------------------//
const int MAX_BOUNCES = 3;
#define FLT_MAX          3.402823466e+38F        // max value
#define M_PI 3.14159265f
//--------------------------------------------------------------------//
__device__ float3 Vec3TransformCoord(float3 in, float m[4][4])
{
    float3 out;
    out.x = in.x * m[0][0] + in.y * m[1][0] + in.z * m[2][0] + 1 * m[3][0];
    out.y = in.x * m[0][1] + in.y * m[1][1] + in.z * m[2][1] + 1 * m[3][1];
    out.z = in.x * m[0][2] + in.y * m[1][2] + in.z * m[2][2] + 1 * m[3][2];
    return out;
};
//--------------------------------------------------------------------//
__device__ float3 Vec3Norm(float3 in)
{
    float3 out = in;
    float k = 1.0 / sqrt(in.x * in.x + in.y * in.y + in.z * in.z);
    out.x *= k; out.y *= k; out.z *= k;
    return out;
}
//--------------------------------------------------------------------//
__device__ float3 Vec3Subtract(float3 in1, float3 in2)
{
    float3 out;
    out.x = in1.x - in2.x;
    out.y = in1.y - in2.y;
    out.z = in1.z - in2.z;
    return out;
}
//--------------------------------------------------------------------//
__device__ float3 Vec3Add(float3 in1, float3 in2)
{
    float3 out;
    out.x = in1.x + in2.x;
    out.y = in1.y + in2.y;
    out.z = in1.z + in2.z;
    return out;
}
//--------------------------------------------------------------------//
__device__ float3 Vec3MultScalar(float3 in, float scalar)
{
    float3 out;
    out.x = in.x * scalar;
    out.y = in.y * scalar;
    out.z = in.z * scalar;
    return out;
}
//--------------------------------------------------------------------//
__device__ float3 Vec3DivScalar(float3 in, float scalar)
{
    float3 out;
    out.x = in.x / scalar;
    out.y = in.y / scalar;
    out.z = in.z / scalar;
    return out;
}
//--------------------------------------------------------------------//
__device__ float Vec3Dot(float3 v1, float3 v2)
{
    return v1.x * v2.x + v1.y * v2.y + v1.z * v2.z;
}
//--------------------------------------------------------------------//
__device__ float3 Vec3Cross(float3 v1, float3 v2)
{
    float3 out;
    out.x =  v1.y * v2.z - v1.z * v2.y;
    out.y = -v1.x * v2.z + v1.z * v2.x;
    out.z =  v1.x * v2.y - v1.y * v2.x;
    return out;
}
//--------------------------------------------------------------------//
__device__ float3 Vec3TransformNormal(float3 in, float m[4][4])
{
    in = Vec3Norm(in);
    float3 out;
    out.x = in.x * m[0][0] + in.y * m[1][0] + in.z * m[2][0];
    out.y = in.x * m[0][1] + in.y * m[1][1] + in.z * m[2][1];
    out.z = in.x * m[0][2] + in.y * m[1][2] + in.z * m[2][2];
    in = Vec3Norm(in);
    return out;
};
//--------------------------------------------------------------------//
__device__ bool IntersectRayAxisAlignedBox(float3& rayOrigin, float3& rayDir, float3& bbmax, float3& bbmin)
{
    //largely copied from the link below
    //https://gamedev.stackexchange.com/questions/18436/most-efficient-aabb-vs-ray-collision-algorithms/18459#18459
     
    // float t = 0;//distance
    float3 dirfrac;
    dirfrac.x = 1.0f / rayDir.x;
    dirfrac.y = 1.0f / rayDir.y;
    dirfrac.z = 1.0f / rayDir.z;
    // lb is the corner of AABB with minimal coordinates - left bottom, rt is maximal corner
    // r.org is origin of ray
    float t1 = (bbmin.x - rayOrigin.x) * dirfrac.x;
    float t2 = (bbmax.x - rayOrigin.x) * dirfrac.x;
    float t3 = (bbmin.y - rayOrigin.y) * dirfrac.y;
    float t4 = (bbmax.y - rayOrigin.y) * dirfrac.y;
    float t5 = (bbmin.z - rayOrigin.z) * dirfrac.z;
    float t6 = (bbmax.z - rayOrigin.z) * dirfrac.z;

    float tmin = max(max(min(t1, t2), min(t3, t4)), min(t5, t6));
    float tmax = min(min(max(t1, t2), max(t3, t4)), max(t5, t6));

    // if tmax < 0, ray (line) is intersecting AABB, but the whole AABB is behind us
    if (tmax < 0)
    {
       // t = tmax;
        return false;
    }

    // if tmin > tmax, ray doesn't intersect AABB
    if (tmin > tmax)
    {
       // t = tmax;
        return false;
    }

  //  t = tmin;
    return true;
}
//-----------------------------------------------------------------------// 
__device__ void CreateCoordinateSystem(float3& N, float3& Nt, float3& Nb)
{
    //https://www.scratchapixel.com/code.php?id=34&origin=/lessons/3d-basic-rendering/global-illumination-path-tracing
    if (abs(N.x) > abs(N.y)) {
        float3 t = { N.z, 0, -N.x };
        Nt = Vec3DivScalar(t, sqrt(N.x * N.x + N.z * N.z));
    }
    else {
        float3 t = { 0, -N.z, N.y };
        Nt = Vec3DivScalar(t, sqrt(N.y * N.y + N.z * N.z));
    }
    Nb = Vec3Cross(N, Nt);
}
//--------------------------------------------------------------------//
__device__ bool IntersectRayTriangle(float3 rayOrigin, float3 rayDir, float3 v0, float3 v1, float3 v2, float* pDist, float* pU, float* pV)
{
    //https://www.scratchapixel.com/lessons/3d-basic-rendering/ray-tracing-rendering-a-triangle/moller-trumbore-ray-triangle-intersection
    //Largely based on the XNA  implemention (see CPU version)
    float kEpsilon = 0.00001f;
    float3 e1 = Vec3Subtract(v1, v0);
    float3 e2 = Vec3Subtract(v2, v0);
    float3 p = Vec3Cross(rayDir, e2);
    float det = Vec3Dot(e1, p);
    if (det >= kEpsilon) {

        float3 s = Vec3Subtract(rayOrigin, v0);
        *pU = Vec3Dot(s, p);
        if (*pU < 0.0 || *pU > det) return false;

        float3 q = Vec3Cross(s, e1);
        *pV = Vec3Dot(rayDir, q);
        if (*pV < 0.0 || *pU + *pV > det) return false;
        *pDist = Vec3Dot(e2, q);
        if (*pDist < 0) return false;
    }
    else if (det <= -kEpsilon) {

        float3 s = Vec3Subtract(rayOrigin, v0);
        *pU = Vec3Dot(s, p);
        if (*pU > 0.0 || *pU < det) return false;

        float3 q = Vec3Cross(s, e1);
        *pV = Vec3Dot(rayDir, q);
        if (*pV > 0.0 || *pU + *pV < det) return false;
        *pDist = Vec3Dot(e2, q);
        if (*pDist > 0) return false;
    }
    else
    {
        return false;
    }
    float invDet = 1.0f / det;
    *pU = *pU * invDet;
    *pV = *pV * invDet;
    *pDist = *pDist * invDet;
    return true;
}
//--------------------------------------------------------------------//
__device__ bool Intersect(CUDAMesh& mesh, float3& rayOrigin, float3& rayDir, float& u, float& v, float& dist, float3& hitpoint, float3& nml)
{
    if (!IntersectRayAxisAlignedBox(rayOrigin, rayDir, mesh.bbmax, mesh.bbmin))
    {
        return false;
    }
  
    if (mesh.pMesh)
    {
        bool bHit = false;
        for (int h = 0; h < 8; h++) { 
            if (Intersect(mesh.pMesh[h], rayOrigin, rayDir, u, v, dist, hitpoint, nml)) {
                //return true;
                bHit = true;
            }
        }
        return bHit;
    }
    bool bHit = false;
    for (int k = 0; k < mesh.nNumTriangles; k++)
    {
        float t = 0.0f;
        FLOAT ua = 0;
        FLOAT va = 0;
        // We have to iterate over all the triangles in order to find the nearest intersection.
        float3 v1 = mesh.pVertices[3 * k + 0].pos;
        float3 v2 = mesh.pVertices[3 * k + 1].pos;
        float3 v3 = mesh.pVertices[3 * k + 2].pos;
        if (IntersectRayTriangle(rayOrigin, rayDir, v1, v2, v3, &t, &ua, &va))
        {
            if (t < dist) {
                dist = t;
                u = (1 - ua - va) * mesh.pVertices[3 * k + 0].tex.x + ua * mesh.pVertices[3 * k + 1].tex.x + va * mesh.pVertices[3 * k + 2].tex.x;
                v = (1 - ua - va) * mesh.pVertices[3 * k + 0].tex.y + ua * mesh.pVertices[3 * k + 1].tex.y + va * mesh.pVertices[3 * k + 2].tex.y;

                hitpoint.x = (1 - ua - va) * mesh.pVertices[3 * k + 0].pos.x + ua * mesh.pVertices[3 * k + 1].pos.x + va * mesh.pVertices[3 * k + 2].pos.x;
                hitpoint.y = (1 - ua - va) * mesh.pVertices[3 * k + 0].pos.y + ua * mesh.pVertices[3 * k + 1].pos.y + va * mesh.pVertices[3 * k + 2].pos.y;
                hitpoint.z = (1 - ua - va) * mesh.pVertices[3 * k + 0].pos.z + ua * mesh.pVertices[3 * k + 1].pos.z + va * mesh.pVertices[3 * k + 2].pos.z;

                nml.x = (mesh.pVertices[3 * k + 0].normal.x + mesh.pVertices[3 * k + 1].normal.x + mesh.pVertices[3 * k + 2].normal.x) / 3;
                nml.y = (mesh.pVertices[3 * k + 0].normal.y + mesh.pVertices[3 * k + 1].normal.y + mesh.pVertices[3 * k + 2].normal.y) / 3;
                nml.z = (mesh.pVertices[3 * k + 0].normal.z + mesh.pVertices[3 * k + 1].normal.z + mesh.pVertices[3 * k + 2].normal.z) / 3;

                nml = Vec3Norm(nml);
                bHit = true;
            }
        }
    } 
    return bHit;
}
//--------------------------------------------------------------------//
__device__ bool TraceRays(CUDAMesh* pMesh, long nNumMeshs, float3& rayOrigin, float3& rayDir, bool bHitOnly, bool bUseTextures, float3& hitpoint, float3& nml, float3& rgb, float& nHitDist)
{
    float tmin = FLT_MAX;
    float u = 0;
    float v = 0;
    long nTexWidth = 0;
    long nTexHeight = 0;
    float3* pTexData = 0;
    float3 diffuse = {0, 0, 0};
    bool bHit = false;
    float dist = FLT_MAX;
    for (int h = 0; h < nNumMeshs; h++)
    {
        if (pMesh[h].bLight) {
            continue;
        }
        float ua = 0;
        float va = 0;  
        if (!Intersect(pMesh[h], rayOrigin, rayDir, ua, va, dist, hitpoint, nml)) {
            continue;
        }
        bHit = true;
        u = ua;
        v = va;
        nTexWidth = 0;
        nTexHeight = 0;
        pTexData = 0;
        if (pMesh[h].pMaterial)
        {
            nTexWidth = pMesh[h].pMaterial->nWidth;
            nTexHeight = pMesh[h].pMaterial->nHeight;
            pTexData = pMesh[h].pMaterial->pTexData;
            diffuse = pMesh[h].pMaterial->diffuse;
        }
    }
    if (bHit)
    {
        if (bHitOnly) return true;

        //Get surface properties (texture and uv...).
        /*if (v >= 1) { v = fmod(v, 1); }
        if (u >= 1) { u = fmod(u, 1); }*/
        while (u > 1) { u = u - 1; }
        while (v > 1) { v = v - 1; }
        while (u < 0) { u = u + 1; }
        while (v < 0) { v = v + 1; }
        long j = nTexWidth * v;
        long h = nTexHeight * u;
        if (!bUseTextures)
        {
            //gray/clay model
            rgb.x = 0.5f;
            rgb.y = 0.5f;
            rgb.z = 0.5f;
        }
        else
        {
            //use the textures
            rgb.x = 0.5f;
            rgb.y = 0.5f;
            rgb.z = 0.5f;
            if (nTexWidth > 0 && pTexData) { 
                long nIndex = h * nTexWidth + j;
                if (nIndex >= 0 && nIndex < nTexWidth * nTexHeight)
                {
                    rgb.x = (float)(pTexData[nIndex].x) / 255.0f;
                    rgb.y = (float)(pTexData[nIndex].y) / 255.0f;
                    rgb.z = (float)(pTexData[nIndex].z) / 255.0f;
                }
            }
            else
            {
                rgb = diffuse;
            }
        }
    }
    return bHit;
}
//-----------------------------------------------------------------------// 
__device__ float3 uniformSampleHemisphere(const float& r1, const float& r2)
{ 
    //https://www.scratchapixel.com/code.php?id=34&origin=/lessons/3d-basic-rendering/global-illumination-path-tracing
    float sinTheta = sqrtf(1 - r1 * r1);
    float phi = 2 * M_PI * r2;
    float x = sinTheta * cos(phi);
    float z = sinTheta * sin(phi);
    float3 out = { x, r1, z };
    return out;
}
//--------------------------------------------------------------------//
__device__ float3 Radiance(long nNumSamples, hiprandState& s, CUDAMesh* pVB, long nNumMeshs, float3& rayOrigin, float3& rayDir, const int& depth, unsigned short* Xi, 
    bool bGlobalIllumination, float3 nSunPos, float3 nSunDir, float nSunIntensity, bool bUseTextures)
{ 
    float3 rgb = { 0, 0, 0 };
    float3 nml = { 0, 0, 0 };
    float3 hitpoint = { 0, 0, 0 };
    if (depth > MAX_BOUNCES) return rgb;
    bool bHit = false;
    float tmin = FLT_MAX;
    float dist = FLT_MAX;
    float nHitDist = 0;
    float3 directLighting = { 0, 0, 0 };
    float3 indirectLighting = { 0, 0, 0 };
    bHit = TraceRays(pVB, nNumMeshs, rayOrigin, rayDir, false, bUseTextures, hitpoint, nml, rgb, nHitDist);

    if (bHit)
    {
        //assume diffuse material
		//Direct light - cast shadow ray towards sun
        float3 hitPoint = hitpoint;
        float3 hitNml = nml;
        float3 sunPos = nSunPos;
        float3 sunDir = Vec3Subtract(sunPos, hitPoint);// sunPos - hitPoint;
        sunDir = Vec3Norm(sunDir);

        float bias = 0.01f;
        float3 hitsunnml = { 0, 0, 0 };
        float3 sunhitpoint = { 0, 0, 0 };
        float3 hitrgb = { 0, 0, 0 };
        float3 rorigin = Vec3Add(hitPoint, Vec3MultScalar(hitNml, bias));
        bHit = TraceRays(pVB, nNumMeshs, rorigin, sunDir, true, bUseTextures, sunhitpoint, hitsunnml, hitrgb, nHitDist);
        if (!bHit)
        { 
            float dp = Vec3Dot(hitNml, sunDir);
            directLighting.x = nSunIntensity * max(0.0f, dp);
            directLighting.y = nSunIntensity * max(0.0f, dp);
            directLighting.z = nSunIntensity * max(0.0f, dp);
        }
        if (bGlobalIllumination)
        {
            //https://www.scratchapixel.com/code.php?id=34&origin=/lessons/3d-basic-rendering/global-illumination-path-tracing
            int N = nNumSamples;
            float3 Nt = { 0, 0, 0 };
            float3 Nb = { 0, 0, 0 }; 
            CreateCoordinateSystem(hitNml, Nt, Nb);
            float pdf = 1 / (2 * M_PI);
            for (int n = 0; n < N; ++n) {
                float r1 = hiprand_uniform(&s);
                float r2 = hiprand_uniform(&s);
                float3 sample = uniformSampleHemisphere(r1, r2);
                float3 sampleWorld = 
                {
                    sample.x * Nb.x + sample.y * hitNml.x + sample.z * Nt.x,
                    sample.x * Nb.y + sample.y * hitNml.y + sample.z * Nt.y,
                    sample.x * Nb.z + sample.y * hitNml.z + sample.z * Nt.z 
                };
                float3 ray = Vec3Add(hitPoint, Vec3MultScalar(sampleWorld, bias));
                //set number of samples to 1 for 2nd, 4rd... bounces
                float3 rd = Radiance(1, s, pVB, nNumMeshs, ray, sampleWorld,
                    depth + 1, Xi, bGlobalIllumination, nSunPos, nSunDir, nSunIntensity, bUseTextures);
                indirectLighting.x = indirectLighting.x + r1 * rd.x / pdf;
                indirectLighting.y = indirectLighting.y + r1 * rd.y / pdf;
                indirectLighting.z = indirectLighting.z + r1 * rd.z / pdf; 
            }
            indirectLighting = Vec3DivScalar(indirectLighting, N);
        }
        //check - not sure if I am meant to multiply indirectLighting by sunintensity - but the images look better...
        rgb.x = (directLighting.x + nSunIntensity * indirectLighting.x) * rgb.x / M_PI;
        rgb.y = (directLighting.y + nSunIntensity * indirectLighting.y) * rgb.y / M_PI;
        rgb.z = (directLighting.z + nSunIntensity * indirectLighting.z) * rgb.z / M_PI;
    } 
    rgb.x = min(rgb.x, 1.0f);
    rgb.y = min(rgb.y, 1.0f);
    rgb.z = min(rgb.z, 1.0f);
    return rgb;
}
//--------------------------------------------------------------------//
void CCUDAPathTracer::CopyMesh(CUDAMesh* pDst, CUDAMesh* pSrc, thrust::host_vector<CCUDAVertex*>& vVertexBuffers, thrust::host_vector<CUDAMesh*>& vMeshBuffers)
{ 
    hipError_t cudaStatus;
    if (pSrc->pMesh)
    {
        CUDAMesh* pChildMesh = 0;
        hipError_t cudaStatus = hipMalloc((void**)&pChildMesh, 8 * sizeof(CUDAMesh));//allocate memory on device for child meshes
        if (cudaStatus == hipSuccess) {
            vMeshBuffers.push_back(pChildMesh);
            for (int h = 0; h < 8; h++)
            {
                CopyMesh(&pChildMesh[h], &(pSrc->pMesh[h]), vVertexBuffers, vMeshBuffers);
            }
            cudaStatus = hipMemcpy(&(pDst->pMesh), &(pChildMesh), sizeof(CUDAMesh*), hipMemcpyHostToDevice);//cleanup pointer
        }
    }
    
    CCUDAVertex* pVert = 0;
    if (pSrc->nNumTriangles > 0)
    {
        cudaStatus = hipMalloc((void**)&pVert, 3 * pSrc->nNumTriangles * sizeof(CCUDAVertex));//allocate memory for vertices on device
        if (cudaStatus == hipSuccess) {
            vVertexBuffers.push_back(pVert);
            cudaStatus = hipMemcpy(pVert, pSrc->pVertices, 3 * pSrc->nNumTriangles * sizeof(CCUDAVertex), hipMemcpyHostToDevice);//copy array of vertices to device
        }
    }
    cudaStatus = hipMemcpy(&(pDst->pVertices), &pVert, sizeof(CCUDAVertex*), hipMemcpyHostToDevice);//cleanup pointer (which may be zero if no triangles);

    cudaStatus = hipMemcpy(&(pDst->bbmin), &(pSrc->bbmin), sizeof(float3), hipMemcpyHostToDevice);
    cudaStatus = hipMemcpy(&(pDst->bbmax), &(pSrc->bbmax), sizeof(float3), hipMemcpyHostToDevice);
    cudaStatus = hipMemcpy(&(pDst->nNumTriangles), &(pSrc->nNumTriangles), sizeof(long), hipMemcpyHostToDevice);
    cudaStatus = hipMemcpy(&(pDst->bLight), &(pSrc->bLight), sizeof(bool), hipMemcpyHostToDevice);
    cudaStatus = hipMemcpy(&(pDst->BB), &(pSrc->BB), sizeof(CCUDAAxisAlignedBox), hipMemcpyHostToDevice);
}
//--------------------------------------------------------------------//
__global__ void PTKernel(volatile int* progress, float3* pOutout, long nClientWidth, long nClientHeight, long nNumSamples, long nDiv, float P0, float P1, float* pToLocal,
    float3 nSunPos, float3 nSunDir, float nSunIntensity, bool bGlobalIllumination, bool bUseTextures, CUDAMesh* pVB, long nNumMeshs)
{
    int threadsPerBlock = blockDim.x * blockDim.y * blockDim.z;
    int threadPosInBlock = threadIdx.x +
        blockDim.x * threadIdx.y +
        blockDim.x * blockDim.y * threadIdx.z;
    int blockPosInGrid = blockIdx.x +
        gridDim.x * blockIdx.y +
        gridDim.x * gridDim.y * blockIdx.z;
    int tid = blockPosInGrid * threadsPerBlock + threadPosInBlock;//calculate global index to array
    long nImageWidth = nClientWidth / nDiv;
    long nImageHeight = nClientHeight / nDiv;
    if (tid < nImageWidth * nImageHeight)
    {
        hiprandState s;
        // seed a random number generator
        hiprand_init(tid * tid, 0, 0, &s);
        int j = tid % nImageWidth;//width
        int h = tid / nImageWidth;//height

        float vx = (+2.0f * (float)nDiv * j / (float)nClientWidth - 1.0f) / P0;// P(0, 0);
        float vy = (-2.0f * (float)nDiv * h / (float)nClientHeight + 1.0f) / P1;// P(1, 1);

        float3 rayOrigin = { 0.0f, 0.0f, 0.0f };
        float3 rayDir = { vx, vy, 1.0f };

        float ToLocal[4][4];
        for (int w = 0; w < 4; w++)
        {
            for (int q = 0; q < 4; q++)
            {
                ToLocal[w][q] = pToLocal[w * 4 + q];
            }
        }
        rayDir = Vec3Norm(rayDir);
        rayOrigin = Vec3TransformCoord(rayOrigin, ToLocal);
        rayDir = Vec3TransformNormal(rayDir, ToLocal);
        rayDir = Vec3Norm(rayDir);
        unsigned short Xi[3] = { 0, 0, (short)j * (short)j * (short)j }; // *** Moved outside for VS2012
        int depth = 0;
        float3 rgb = Radiance(nNumSamples, s, pVB, nNumMeshs, rayOrigin, rayDir, depth, Xi, bGlobalIllumination, nSunPos, nSunDir, nSunIntensity, bUseTextures);

        atomicAdd((int*)progress, 1);//increment the progress count that will be propagated back to the progress bar in the UI

        if (rgb.x > 0.1)
        {
            long nStop = 0;
        }
        pOutout[tid].x = 255 * rgb.x;
        pOutout[tid].y = 255 * rgb.y;
        pOutout[tid].z = 255 * rgb.z;
    }
}
//--------------------------------------------------------------------//
void CCUDAPathTracer::CalcRays(CPTCallback* pCallback, float3* pOutputImage, long nClientWidth, long nClientHeight, long nNumSamples, long nDiv, float P0, float P1, float ToLocal[4][4],
                           float3 nSunPos, float3 nSunDir, float nSunIntensity, bool bGlobalIllumination, bool bUseTextures, CUDAMesh* pVB, long nNumMeshs, CUDAMaterial* pMaterials, long nNumMaterials)
{
    long nWidth = nClientWidth / nDiv;
    long nHeight = nClientHeight / nDiv;
    thrust::host_vector<CCUDAVertex*> vVertexBuffers;
    thrust::host_vector<CUDAMesh*> vMeshBuffers;
    thrust::host_vector<float3*> vTextures;

    thrust::host_vector<CUDAMaterial*> vHostMaterials;
    thrust::host_vector<CUDAMaterial*> vDeviceMaterials;
  
   // short* devHGTData = 0;
    float3* pCUDAOutputImage = 0;
    float* pToLocal = 0;
    hipError_t cudaStatus;    
    CUDAMesh* pCUDAVB = 0;
    CUDAMaterial* pCUDAMaterials = 0;

    ////////////////////////////////////////////////////////////////////////////////////////////////
    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }
  
    ////////////////////////////////////////////////////////////////////////////////////////////////
    // Allocate GPU buffers for output image
    cudaStatus = hipMalloc((void**)&pCUDAOutputImage, nWidth * nHeight * sizeof(float3));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
      
    ////////////////////////////////////////////////////////////////////////////////////////////////
    //Copy materials/textures
    cudaStatus = hipMalloc((void**)&pCUDAMaterials, nNumMaterials * sizeof(CUDAMaterial));
    if (cudaStatus == hipSuccess) {
        for (int h = 0; h < nNumMaterials; h++) {
            vDeviceMaterials.push_back(&pCUDAMaterials[h]);//store the host and device version of this structure so we can map them later
            vHostMaterials.push_back(&pMaterials[h]);
            float3* pTexData = 0;
            cudaStatus = hipMalloc((void**)&pTexData, pMaterials[h].nWidth * pMaterials[h].nHeight * sizeof(float3));
            if (cudaStatus != hipSuccess) {
                fprintf(stderr, "hipMalloc failed!");
                goto Error;
            }
            
            cudaStatus = hipMemcpy(&pCUDAMaterials[h].nWidth, &pMaterials[h].nWidth, sizeof(long), hipMemcpyHostToDevice);
            cudaStatus = hipMemcpy(&pCUDAMaterials[h].nHeight, &pMaterials[h].nHeight, sizeof(long), hipMemcpyHostToDevice);
            cudaStatus = hipMemcpy(pTexData, pMaterials[h].pTexData, pMaterials[h].nWidth * pMaterials[h].nHeight * sizeof(float3), hipMemcpyHostToDevice);
            cudaStatus = hipMemcpy(&pCUDAMaterials[h].diffuse, &pMaterials[h].diffuse, sizeof(float3), hipMemcpyHostToDevice);
            cudaStatus = hipMemcpy(&(pCUDAMaterials[h].pTexData), &pTexData, sizeof(float3*), hipMemcpyHostToDevice);
            //cudaStatus = hipMemcpy(&(pDst->pMesh), &(pChildMesh), sizeof(CUDAMesh*), hipMemcpyHostToDevice);//cleanup pointer
       
            vTextures.push_back(pTexData);//store in a vector so we can delete them later
        }
    }
    
    ////////////////////////////////////////////////////////////////////////////////////////////////
    // Allocate GPU buffers for 4 by 4 matrix
    cudaStatus = hipMalloc((void**)&pToLocal, 16 * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    {
        //copy 4 by 4 matrix to device
        float* pTemp = new float[16];
        for (int w = 0; w < 4; w++) {
            for (int q = 0; q < 4; q++) {
                pTemp[w * 4 + q] = ToLocal[w][q];
            }
        }
        hipError_t cudaStatus = hipMemcpy(pToLocal, pTemp, 16 * sizeof(float), hipMemcpyHostToDevice);
        delete[] pTemp;
    }

    ////////////////////////////////////////////////////////////////////////////////////////////////
    //Increase stack size otherwise the recursion will generate a 719 error
    cudaStatus = cudaThreadSetLimit(hipLimitStackSize, 12000);
    // Allocate GPU buffers for mesh array
    cudaStatus = hipMalloc((void**)&pCUDAVB, nNumMeshs * sizeof(CUDAMesh));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    
    ////////////////////////////////////////////////////////////////////////////////////////////////
    //Copy meshes
    cudaStatus = hipMemset(pCUDAVB, 0, nNumMeshs * sizeof(CUDAMesh));
    for (int h = 0; h < nNumMeshs; h++)
    {        
        CopyMesh(&pCUDAVB[h], &pVB[h], vVertexBuffers, vMeshBuffers);
        //replace pointer to host material to pointer to the same material but on the device (that we computed a moment ago)
        if (vHostMaterials.size() != vDeviceMaterials.size()) {
            long nStop = 0;
        }
        for (int j = 0; j < vHostMaterials.size(); j++)
        {
            CUDAMaterial* pTest = vHostMaterials[j];
            if (pVB[h].pMaterial == pTest)
            {
                cudaStatus = hipMemcpy(&pCUDAVB[h].pMaterial, &vDeviceMaterials[j], sizeof(CUDAMaterial*), hipMemcpyHostToDevice);
                break;
            }            
        }
    }
     
    ////////////////////////////////////////////////////////////////////////////////////////////////
    //Used to send progress from CUDA device back to host
    volatile int* d_progress, * h_progress;    
    cudaStatus = hipSetDeviceFlags(hipDeviceMapHost);
    cudaStatus = hipHostAlloc((void**)&h_progress, sizeof(int), hipHostMallocMapped);
    cudaStatus = hipHostGetDevicePointer((int**)&d_progress, (int*)h_progress, 0);
    *h_progress = 0;
    *d_progress = 0;

    ////////////////////////////////////////////////////////////////////////////////////////////////
    // Launch a kernel on the GPU with one thread for each element.
    cudaStatus = hipDeviceSynchronize();

    long tx = 8;
    long ty = 8;
    dim3 block(nWidth / tx + 1, nHeight / ty + 1);
    dim3 threads(tx, ty);

    hipEvent_t start, stop;
    hipEventCreate(&start); 
    hipEventCreate(&stop);
    hipEventRecord(start);

    ////////////////////////////////////////////////////////////////////////////////////////////////
    //The Kernel
    PTKernel << <block, threads >> > (d_progress, pCUDAOutputImage, nClientWidth, nClientHeight, nNumSamples, nDiv, P0, P1, pToLocal, nSunPos, nSunDir, nSunIntensity, bGlobalIllumination, bUseTextures, pCUDAVB, nNumMeshs);

    hipEventRecord(stop);
  
    ////////////////////////////////////////////////////////////////////////////////////////////////
    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "HGTToNormalKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    ////////////////////////////////////////////////////////////////////////////////////////////////
    //send progress back to caller
    if (pCallback)
    {
        pCallback->UpdateProgress(0, nWidth * nHeight);
        //https://stackoverflow.com/questions/20345702/how-can-i-check-the-progress-of-matrix-multiplication
        int nProgress = 0;
        do {
            hipEventQuery(stop);
            int nProgress1 = (*h_progress) * 100 / nWidth / nHeight;
            if (nProgress1 > nProgress) 
            {
                nProgress = nProgress1;
                pCallback->UpdateProgress( *h_progress, nWidth * nHeight);
            }
        } while (*h_progress < nWidth * nHeight - 2);
    }

    ////////////////////////////////////////////////////////////////////////////////////////////////
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching HGTToNormalKernel!\n", cudaStatus);
        goto Error;
    }

    ////////////////////////////////////////////////////////////////////////////////////////////////
    // Copy output image from GPU buffer to host memory.
    cudaStatus = hipMemcpy(pOutputImage, pCUDAOutputImage, nWidth * nHeight * sizeof(float3), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    ////////////////////////////////////////////////////////////////////////////////////////////////
    //Free Memory
    hipFree(pCUDAOutputImage);
    for(auto& i : vVertexBuffers)
    {
        hipFree(i);
    }
    for (auto& i : vMeshBuffers)
    {
        hipFree(i);
    }
    for (auto& i : vTextures)
    {
        hipFree(i);
    }
    hipFree(pCUDAMaterials);
    hipFree(pCUDAVB);
    hipFree(pToLocal);
}
//--------------------------------------------------------------------//
